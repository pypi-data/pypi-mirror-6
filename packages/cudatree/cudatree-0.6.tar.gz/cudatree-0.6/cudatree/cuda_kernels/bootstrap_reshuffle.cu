#include "hip/hip_runtime.h"
#include<stdio.h>
#include<math.h>
#include<stdint.h>
#define IDX_DATA_TYPE %s
#define THREADS_PER_BLOCK %s

texture<char, 1> tex_mark;

__global__ void bootstrap_reshuffle(uint8_t* mark_table,
                          IDX_DATA_TYPE* sorted_indices,
                          IDX_DATA_TYPE* sorted_indices_out,
                          uint32_t stride
                          ){  
  uint32_t indices_offset = blockIdx.x * stride;
  IDX_DATA_TYPE reg_pos = 0;
  uint32_t out_pos;
  uint8_t side;
  IDX_DATA_TYPE n;

  __shared__ IDX_DATA_TYPE last_sum;
  __shared__ IDX_DATA_TYPE shared_pos_table[THREADS_PER_BLOCK];
  
  if(threadIdx.x == 0)
    last_sum = 0;
  
  for(IDX_DATA_TYPE i = threadIdx.x; i < stride; i += blockDim.x){
    side = tex1Dfetch(tex_mark, sorted_indices[indices_offset + i]);//mark_table[sorted_indices[indices_offset + i]];
    //side = mark_table[sorted_indices[indices_offset + i]];
    reg_pos = side;
    
    shared_pos_table[threadIdx.x] = reg_pos;

    __syncthreads();
     
    for(uint16_t s = 1; s < blockDim.x; s *= 2){
      if(threadIdx.x >= s){
        n = shared_pos_table[threadIdx.x - s];
      }
      else 
        n = 0;

      __syncthreads();
      shared_pos_table[threadIdx.x] += n;
      __syncthreads();
    }
 
    reg_pos = shared_pos_table[threadIdx.x] + last_sum;

    if(side == 1){
      out_pos = indices_offset + reg_pos - 1;
      sorted_indices_out[out_pos] = sorted_indices[indices_offset + i];   
    }
    
    __syncthreads();
    
    if(threadIdx.x == blockDim.x - 1)
      last_sum = reg_pos; 
  }
}

